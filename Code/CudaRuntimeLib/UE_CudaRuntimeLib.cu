#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "cuda_surface_types.h"
#include "hip/hip_runtime_api.h"
#include "surface_functions.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


// CUDA surface kernel: write solid red into the texture
//__global__ void FillSurfaceKernel(hipSurfaceObject_t surf, int width, int height)
//{
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if (x < width && y < height)
//    {
//        uchar4 pixel = make_uchar4(255, 0, 255, 255); // RGBA8 red
//        surf2Dwrite(pixel, surf, x * sizeof(uchar4), y);
//    }
//}
__global__ void FillSurfaceKernel(hipSurfaceObject_t surf, int width, int height, unsigned int seed)
{
	//int seed = 0; // You can modify this seed for different patterns
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Simple hash function for pseudo randomness
        unsigned int value = (x * 1973 + y * 9277 + seed * 26699) | 1;
        value ^= value << 13;
        value ^= value >> 17;
        value ^= value << 5;

        // Extract channels
        unsigned char r = (value & 0xFF);
        unsigned char g = (value >> 8) & 0xFF;
        unsigned char b = (value >> 16) & 0xFF;

        uchar4 pixel = make_uchar4(r, g, b, 255);
        surf2Dwrite(pixel, surf, x * sizeof(uchar4), y);
    }
}


// C wrapper function (called from Unreal)
hipError_t LaunchFillSurfaceKernel(hipArray_t array, int width, int height, unsigned int seed)
{
    hipError_t cudaStatus = hipSuccess;

    // Describe the surface
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    // Create surface object
    hipSurfaceObject_t surfObj = 0;
    cudaStatus = hipCreateSurfaceObject(&surfObj, &resDesc);
    if (cudaStatus != hipSuccess)
        return cudaStatus;

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
        (height + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    FillSurfaceKernel << <gridDim, blockDim >> > (surfObj, width, height, seed);

    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        hipDestroySurfaceObject(surfObj);
        return cudaStatus;
    }

    // Wait for kernel to finish and check for runtime errors
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        hipDestroySurfaceObject(surfObj);
        return cudaStatus;
    }

    // Destroy surface object
    cudaStatus = hipDestroySurfaceObject(surfObj);
    return cudaStatus;
}